#include "hip/hip_runtime.h"

#include "VectorAdd.h"
#include "CudaUtil.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>


__global__ void vectorAddKernel(float* inputA, float* inputB, float* output, int length){

	//compute element index
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//add an vector element
	if(idx < length) output[idx] = inputA[idx] + inputB[idx];

}


void vectorAddHost(const float* A, const float* B, float* output, int length){

	float *devA, *devB, *devC;
	int size = length * sizeof(float);

	//allocate device memory
	CUDA_CHECK(hipMalloc(&devA, size));
	CUDA_CHECK(hipMalloc(&devB, size));
	CUDA_CHECK(hipMalloc(&devC, size));

	//copy values from host to device
	CUDA_CHECK(hipMemcpy(devA, A, size, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(devB, B, size, hipMemcpyHostToDevice));

	//lunch kernel
	dim3 blockDim(256, 1, 1);
	dim3 gridDim((int)ceil(length/(WARP_SIZE*1.0)), 1, 1);
	vectorAddKernel<<<gridDim,blockDim>>>(devA, devB, devC, length);

	//synchronize with host with async kernel
	CUDA_CHECK(hipDeviceSynchronize());

	//copy values from device back to host
	CUDA_CHECK(hipMemcpy(output, devC, size, hipMemcpyDeviceToHost));

	//free device memory
	CUDA_CHECK(hipFree(devA));
	CUDA_CHECK(hipFree(devB));
	CUDA_CHECK(hipFree(devC));
}
