#include "hip/hip_runtime.h"
#include "MatrixVectorMult.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "CudaUtil.h"
#include <math.h>
#include <stdio.h>

__global__ void matrixVectorMultKernel(float* fltMatrix, float* vec, float* output, int rows, int columns){

	int row = blockDim.x * blockIdx.x + threadIdx.x;

	if(row < rows){
		float sum = 0.0f;
		for (int col = 0; col < columns; ++col) {
			sum += fltMatrix[row * columns + col] + vec[col];
		}

		output[row] = sum;
	}

}

void MatrixVectorMultHost(const float* fltMatA, const float* vecB, float *output, int rows, int columns){

	float *devFltMat, *devVecB, *devOutput;
	int matEls = rows * columns;
	int vecEls = columns;
	int outPutEls = rows;
	int matSize = matEls * sizeof(float);
	int vecSize = vecEls * sizeof(float);
	int outputSize = outPutEls * sizeof(float);

	//Allocate memory on GPU
	CUDA_CHECK(hipMalloc(&devFltMat, matSize));
	CUDA_CHECK(hipMalloc(&devVecB, vecSize));
	CUDA_CHECK(hipMalloc(&devOutput, outputSize));

	//Copy memory to GPU
	CUDA_CHECK(hipMemcpy(devFltMat,fltMatA,matSize,hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(devVecB,vecB,vecSize,hipMemcpyHostToDevice));

	//configure, lunch and synchronize kernel
	dim3 blockDim(256, 1, 1);
	dim3 gridDim(ceil(rows/256.0f), 1, 1);
	matrixVectorMultKernel<<<gridDim,blockDim>>>(devFltMat,devVecB,devOutput,rows,columns);
	CUDA_CHECK(hipDeviceSynchronize());

	//copy memory back to host
	CUDA_CHECK(hipMemcpy(output, devOutput, outputSize, hipMemcpyDeviceToHost));

	//free device memory
	CUDA_CHECK(hipFree(devFltMat));
	CUDA_CHECK(hipFree(devVecB));
	CUDA_CHECK(hipFree(devOutput));

}
