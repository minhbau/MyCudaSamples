#include "hip/hip_runtime.h"
#include "VectorReduction.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__constant__ float mask_cte[3];

__global__ void conv1D(float* dev_in, int input_width, int mask_width, float* dev_out){

	//index computation
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < input_width){

		float result = 0.0f;
		//start index on the input array
		int start = idx - (mask_width/2);
		//compute convolution
		for(int j = 0; j<mask_width ; j++ ){
			//boundary check
			if((start + j >= 0) && (start + j < input_width)){
				result += dev_in[start + j] * mask_cte[j];
			}
		}
		//store final value
		dev_out[idx] = result;
	}
}

__global__ void tiledConv1D(float* dev_in, int input_width, int mask_width, float* dev_out){

	//index computation
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//load device memory coperatively
	extern __shared__ float ds_in[];
	int n = mask_width/2;

	//load left halo elements uising the las thread in current block
	int haloIdxLeft = (blockIdx.x-1)*blockDim.x + threadIdx.x;
	if(threadIdx.x >= blockDim.x - n)
		ds_in[threadIdx.x - (blockDim.x - n)] = haloIdxLeft < 0 ? 0 : dev_in[haloIdxLeft];

	//load central elements
	ds_in[n + threadIdx.x]

	//load right halo elements


	if(idx < input_width){

		float result = 0.0f;
		//start index on the input array
		int start = idx - (mask_width/2);
		//compute convolution
		for(int j = 0; j<mask_width ; j++ ){
			//boundary check
			if((start + j >= 0) && (start + j < input_width)){
				result += dev_in[start + j] * mask_cte[j];
			}
		}
		//store final value
		dev_out[idx] = result;
	}
}

__global__ void conv2D(float* dev_in, int input_width, float* dev_mask, int mask_width,float* dev_out){

}


void conv1DHost(const float* input, const int input_width, const float* mask, const int mask_width, float* output){

	//variables
	float *dev_in, *dev_mask, *dev_out;
	int size_InOut = input_width * sizeof(float);
	int size_mask = mask_width * sizeof(float);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mask_cte),mask,size_mask));


	//allocate memory on gpu
	checkCudaErrors(hipMalloc(&dev_in,size_InOut));
	checkCudaErrors(hipMalloc(&dev_out,size_InOut));
	checkCudaErrors(hipMalloc(&dev_mask, size_mask));

	//copy data to gpu
	checkCudaErrors(hipMemcpy(dev_in,input,size_InOut,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_mask,mask,size_mask,hipMemcpyHostToDevice));

	//configure, lunch and synchronize kernel
	dim3 blocks(256,1,1);
	dim3 grid(ceil(input_width/256.0f),1,1);
	conv1D<<<grid,blocks>>>(dev_in,input_width,mask_width,dev_out);
	checkCudaErrors(hipDeviceSynchronize());

	//copy data back to host
	checkCudaErrors(hipMemcpy(output,dev_out,size_InOut,hipMemcpyDeviceToHost));

	//free memory
	checkCudaErrors(hipFree(dev_in));
	checkCudaErrors(hipFree(dev_mask));
	checkCudaErrors(hipFree(dev_out));

}

void conv2DHost(const float* input, const int input_width, const float* mask, const int mask_width, float* output){}
